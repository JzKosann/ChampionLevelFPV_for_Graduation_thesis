#include "hip/hip_runtime.h"
//
// Created by hg on 2024/9/11.
//
//#include "hip/hip_runtime.h"
#include <stdio.h>


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
extern "C"
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

int getThreadNum()
{
    hipDeviceProp_t prop{};
    int count;

    HANDLE_ERROR(hipGetDeviceCount(&count));
    printf("gpu num %d\n", count);
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
    printf("max thread num: %d\n", prop.maxThreadsPerBlock);
    printf("max grid dimensions: %d, %d, %d)\n",
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    return prop.maxThreadsPerBlock;
}

int cuda_test(){
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//           c[0], c[1], c[2], c[3], c[4]);
    printf("cuda template is available！\n");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    getThreadNum();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    int dev;
    for (dev = 0; dev < deviceCount; dev++)
    {
        int driver_version(0), runtime_version(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        if (dev == 0)
            if (deviceProp.minor = 9999 && deviceProp.major == 9999)
                printf("\n");

        printf("\nDevice%d:\"%s\"\n", dev, deviceProp.name);
        hipDriverGetVersion(&driver_version);
        printf("CUDA Driver Version:                            %d.%d\n",
               driver_version / 1000, (driver_version % 1000) / 10);
        hipRuntimeGetVersion(&runtime_version);
        printf("CUDA Runtime Version:                           %d.%d\n",
               runtime_version / 1000, (runtime_version % 1000) / 10);
        printf("Device Prop:                                    %d.%d\n",
               deviceProp.major, deviceProp.minor);
        printf("Total amount of Global Memory:                  %zu bytes\n",
               deviceProp.totalGlobalMem);
        printf("Number of SMs:                                  %d\n",
               deviceProp.multiProcessorCount);
        printf("Total amount of Constant Memory:                %zu bytes\n",
               deviceProp.totalConstMem);
        printf("Total amount of Shared Memory per block:        %zu bytes\n",
               deviceProp.sharedMemPerBlock);
        printf("Total number of registers available per block:  %d\n",
               deviceProp.regsPerBlock);
        printf("Warp size:                                      %d\n",
               deviceProp.warpSize);
        printf("Maximum number of threads per SM:               %d\n",
               deviceProp.maxThreadsPerMultiProcessor);
        printf("Maximum number of threads per block:            %d\n",
               deviceProp.maxThreadsPerBlock);
        printf("Maximum size of each dimension of a block:      %d x %d x %d\n",
               deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf("Maximum size of each dimension of a grid:       %d x %d x %d\n",
               deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        printf("Maximum memory pitch:                           %zu bytes\n",
               deviceProp.memPitch);
        printf("Texture alignmemt:                              %zu bytes\n",
               deviceProp.texturePitchAlignment);
        printf("Clock rate:                                     %.2f GHz\n",
               (float)deviceProp.clockRate * 1e-6f);
        printf("Memory Clock rate:                              %.0f MHz\n",
               (float)deviceProp.memoryClockRate * 1e-3f);
        printf("Memory Bus Width:                               %d-bit\n",
               deviceProp.memoryBusWidth);
    }
    return 0;
}